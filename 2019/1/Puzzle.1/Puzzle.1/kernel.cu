
#include "hip/hip_runtime.h"


#include <iostream>
#include <iomanip>
#include <fstream>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void doCudaComputation(int* input, int* output);
int doComputationOutput2(int input);

void readFile(int* input);

__global__ void addKernel(int *output, const int *input)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int x = input[i] / 3;
	output[i] = x - 2;
}

int main()
{
	int* input = new int[100];
	int* output = new int[100];

	readFile(input);
	doCudaComputation(input, output);

	int sum = 0;

	for (int i = 0; i < 100; ++i) {
		sum += output[i];
	}

	int sum2 = 0;

	for (int i = 0; i < 100; ++i) {
		sum2 += doComputationOutput2(output[i]);
	}

	printf("Sum: %d\n", sum);
	printf("Sum2: %d\n", sum2);

    return 0;
}


int doComputationOutput2(int input) {
	int x = input / 3;

	if (x != 0 && x - 2 > 0) {
		return input + doComputationOutput2(x - 2);
	}
	else {
		return input;
	}
}


void readFile(int* input) {
	int mass = 0;
	int compt = 0;
	std::ifstream inFile;

	inFile.open("input.txt");

	while (inFile >> mass) {
		input[compt] = mass;
		compt++;
	}
}

void doCudaComputation(int *input, int *output) {
	int *dev_input = nullptr;
	int *dev_output = nullptr;
	hipError_t cudaStatus;
	
	hipSetDevice(0);

	cudaStatus = hipMalloc((void**)&dev_input, 100 * sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_output, 100 * sizeof(int));


	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	hipMemcpy(dev_input, input, 100 * sizeof(int), hipMemcpyHostToDevice);

	addKernel<<<1, 100>>>(dev_output, dev_input);
	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, 100 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
}

//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}
