
#include "hip/hip_runtime.h"


#include <iostream>
#include <iomanip>
#include <fstream>

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void doCudaComputation(int* input, int* output);
int doComputationOutput2(int input);

void readFile(int* input);

__global__ void addKernel(int *output, const int *input)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int x = input[i] / 3;
	output[i] = x - 2;
}

int main()
{
	int* input = new int[100];
	int* output = new int[100];

	readFile(input);
	doCudaComputation(input, output);

	int sum = 0;

	for (int i = 0; i < 100; ++i) {
		sum += output[i];
	}

	int sum2 = 0;

	for (int i = 0; i < 100; ++i) {
		sum2 += doComputationOutput2(output[i]);
	}

	printf("Sum: %d\n", sum);
	printf("Sum2: %d\n", sum2);

    return 0;
}


int doComputationOutput2(int input) {
	int x = input / 3;

	if (x != 0 && x - 2 > 0) {
		return input + doComputationOutput2(x - 2);
	}
	else {
		return input;
	}
}


void readFile(int* input) {
	int mass = 0;
	int compt = 0;
	std::ifstream inFile;

	inFile.open("input.txt");

	while (inFile >> mass) {
		input[compt] = mass;
		compt++;
	}
}

void doCudaComputation(int *input, int *output) {
	int *dev_input = nullptr;
	int *dev_output = nullptr;
	hipError_t cudaStatus;
	
	hipSetDevice(0);

	hipMalloc((void**)&dev_input, 100 * sizeof(int));
	hipMalloc((void**)&dev_output, 100 * sizeof(int));

	hipMemcpy(dev_input, input, 100 * sizeof(int), hipMemcpyHostToDevice);

	addKernel<<<1, 100>>>(dev_output, dev_input);
	hipDeviceSynchronize();

	hipMemcpy(output, dev_output, 100 * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_input);
	hipFree(dev_output);
}
